#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "opencv2/highgui/highgui.hpp"
#pragma comment(lib, "cudart")

#include <math.h>
#include <stdio.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include "array.h"
#include "d_cuda.h"
#include "multiple.h"


using std::cerr;
using std::cout;
using std::endl;
using std::exception;
using std::vector;
using std::iterator;


using namespace std;
using namespace cv;
// CUDA kernel: adds each array value

#define size 8

int main(void)
{
	//////////////////////////////// INIT /////////////////////////
	
	matrix_2D A,B,C;		
	unsigned int mem_size_C = sizeof(int) * size*size;
	create_size_matrix_2D(A,size,size);
	create_size_matrix_2D(B,size,size);
	create_size_matrix_2D(C,size,size);

	info_matrix_2D(A);
	rand_matrix_2D(A);
	show_matrix_2D(A);

	info_matrix_2D(B);
	rand_matrix_2D(B);
	show_matrix_2D(B);

	info_matrix_2D(C);
	fill_matrix_2D(C,0);
	//show_matrix_2D(C);
	
	int *da ,*db ,* dc;
	hipMalloc (( void **) & dc , size*size* sizeof ( int ));
	hipMalloc (( void **) & db , size*size* sizeof ( int ));
	hipMalloc (( void **) & da , size*size* sizeof ( int ));

	host_to_device(da,A);
	host_to_device(db,B);

	// setup execution parameters
   int numThreadsPerBlock = size;
   int numBlocks = (size * size);
   int sharedMemSize = numThreadsPerBlock * sizeof(int);

   dim3 dimGrid(numBlocks);
   dim3 dimBlock(numThreadsPerBlock);

   // execute the kernel
   matrixMul <<< dimGrid, dimBlock, sharedMemSize >>>(da, db, dc);
   
   //Block until device has completed
   hipDeviceSynchronize();
   device_to_host(dc,C);
   show_matrix_2D(C);


	hipFree ( dc) ; hipFree ( da); hipFree ( db);

	A.clear();
	B.clear();
	C.clear();

	
  system("PAUSE");    
}