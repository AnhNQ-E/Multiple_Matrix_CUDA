#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "opencv2/highgui/highgui.hpp"
#pragma comment(lib, "cudart")

#include <math.h>
#include <stdio.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include "array.h"
#include "d_cuda.h"
#include "multiple.h"


using std::cerr;
using std::cout;
using std::endl;
using std::exception;
using std::vector;
using std::iterator;


using namespace std;
using namespace cv;
// CUDA kernel: adds each array value

#define size 8
#define col1 8
#define col2 3
#define row1 3
#define row2 8

int main(void)
{
	//////////////////////////////// INIT /////////////////////////
	
	matrix_2D A,B,C;		
	matrix_1D D,E;
	create_size_matrix_2D(A,row1,col1);
	create_size_matrix_2D(B,row2,col2);
	create_size_matrix_2D(C,row1,col2);

	D.resize(row1*col1);
	E.resize(col2*row2);
	
	info_matrix_2D(A);
	rand_matrix_2D(A);
	show_matrix_2D(A);

	info_matrix_2D(B);
	rand_matrix_2D(B);
	show_matrix_2D(B);

	info_matrix_2D(C);
	fill_matrix_2D(C,0);
	show_matrix_2D(C);
	cout<<endl<<endl;
	matrix_2D_to_1D_A(D,A);
	matrix_2D_to_1D_B(E,B);
	for(int i=0; i< row1*col1; i++)
		cout<<D[i]<<" ";
	cout<<endl;
	for(int i=0; i< row2*col2; i++)
		cout<<E[i]<<" ";
	cout<<endl;
	
	int *da ,*db ,* dc;
	hipMalloc (( void **) & da , col1*row1* sizeof ( int ));
	hipMalloc (( void **) & db , col2*row2* sizeof ( int ));
	hipMalloc (( void **) & dc , row1*col2* sizeof ( int ));

	//host_to_device_2D(da,A);
	//host_to_device_2D(db,B);

	// setup execution parameters
	host_to_device_1D(da,D);
	host_to_device_1D(db,E);
	MatrixMulKernel<<<col2,row1>>>(da,db,dc,row2,col2,row1);

   
   //Block until device has completed
   
   device_to_host_2D(dc,C);
   cout<<endl;
   show_matrix_2D(C);
   

	hipFree ( dc) ; hipFree ( da); hipFree ( db);
	
	A.clear();
	B.clear();
	C.clear();

	
  system("PAUSE");    
}